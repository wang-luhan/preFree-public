#include "hip/hip_runtime.h"
#include "common.h"
/*
如果重排行，似乎也没用；因为是以512为粒度

一旦512定了，这个稀疏矩阵中所有tile情况就定了

情况：128-512行; 64-128; 32-64

tile内: bucket
if: LUT
*/

__global__ void pre_startRowPerBlock(const int *__restrict__ row_ptr,
                                     const int m,
                                     int *__restrict__ startRowPerBlock,
                                     int tileSize)
{
  const int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (global_thread_id >= m)
    return;
  int a = row_ptr[global_thread_id];
  int b = row_ptr[min(global_thread_id + 1, (int)m)];
  int blocka = divup<int>(a, tileSize);
  int blockb = (b - 1) / static_cast<int>(tileSize);
  if (a != b)
    for (; blocka <= blockb; ++blocka)
      startRowPerBlock[blocka] = global_thread_id;
}

template <int THREADS_PER_BLOCK>
__device__ __forceinline__ void reduce_oneRow_in_block(int NNZ_PER_BLOCK, const int tid_in_block, const int block_id,
                                                       const int reduceStartRowId,
                                                       const int *__restrict__ row_ptr,
                                                       const valT *__restrict__ smem, valT *__restrict__ y)
{
  __shared__ valT LDS[THREADS_PER_BLOCK];
  // __shared__ volatile valT LDS[THREADS_PER_BLOCK];

  valT sum = 0;
  const int reduce_start_idx = max((int)0, row_ptr[reduceStartRowId] - block_id * NNZ_PER_BLOCK);
  const int reduce_end_idx = min(NNZ_PER_BLOCK, row_ptr[reduceStartRowId + 1] - block_id * NNZ_PER_BLOCK);

  for (int j = reduce_start_idx + threadIdx.x; j < reduce_end_idx; j += THREADS_PER_BLOCK)
  {
    sum += smem[j];
  }
  LDS[threadIdx.x] = sum;
  __syncthreads();

  // Reduce partial sums
  for (int stride = THREADS_PER_BLOCK >> 1; stride > 0; stride >>= 1)
  {
    __syncthreads();
    if (threadIdx.x < stride)
      LDS[threadIdx.x] += LDS[threadIdx.x + stride];
  }
  // Write result
  if (threadIdx.x == 0)
    atomicAdd(y + reduceStartRowId, LDS[threadIdx.x]);
}

template <int THREADS_PER_BLOCK>
__device__ __forceinline__ void reduce_oneRow_in_thread(int NNZ_PER_BLOCK, const int tid_in_block, const int block_id,
                                                        const int reduceStartRowId, const int reduceEndRowId,
                                                        const int *__restrict__ row_ptr,
                                                        const valT *__restrict__ smem, valT *__restrict__ y)
{
  int reduce_row_id = reduceStartRowId + tid_in_block;
  int nnz_id_before = block_id * NNZ_PER_BLOCK;
  for (; reduce_row_id < reduceEndRowId; reduce_row_id += THREADS_PER_BLOCK)
  {
    valT sum = 0;
    const int reduce_start_idx = max((int)0, row_ptr[reduce_row_id] - nnz_id_before);
    const int reduce_end_idx = min(NNZ_PER_BLOCK, row_ptr[reduce_row_id + 1] - nnz_id_before);
    for (int i = reduce_start_idx; i < reduce_end_idx; i++)
    {
      sum += smem[i];
    }
    atomicAdd(y + reduce_row_id, sum);
  }
}

template <int VEC_SIZE>
__device__ __forceinline__ valT warpReduceSum(valT sum)
{
  if (VEC_SIZE >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
  if (VEC_SIZE >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8); // 0-8, 1-9, 2-10, etc.
  if (VEC_SIZE >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4); // 0-4, 1-5, 2-6, etc.
  if (VEC_SIZE >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2); // 0-2, 1-3, 4-6, 5-7, etc.
  if (VEC_SIZE >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1); // 0-1, 2-3, 4-5, etc.
  return sum;
}

template <int THREADS_PER_BLOCK, int VECTOR_SIZE>
__device__ __forceinline__ void
reduce_oneRow_in_vector(int NNZ_PER_BLOCK, const int n_reduce_rows_num, const int tid_in_block, const int block_id,
                        const int reduceStartRowId, const int reduceEndRowId,
                        const int *__restrict__ row_ptr, const valT *__restrict__ smem, valT *__restrict__ y)
{
  // use `vec_num` vectors, each vector can process reduction of one row by involving `vec_size` threads.
  const int vec_size = VECTOR_SIZE;
  const int vec_num = THREADS_PER_BLOCK / vec_size;
  const int vec_id = tid_in_block / vec_size;
  const int tid_in_vec = tid_in_block & (vec_size - 1);

  int reduce_row_id = reduceStartRowId + vec_id;
  for (; reduce_row_id < reduceEndRowId; reduce_row_id += vec_num)
  {
    const int reduce_start_idx = max((int)0, row_ptr[reduce_row_id] - block_id * NNZ_PER_BLOCK);
    const int reduce_end_idx = min((int)NNZ_PER_BLOCK, row_ptr[reduce_row_id + 1] - block_id * NNZ_PER_BLOCK);
    // reduce LDS via vectors.
    valT sum = 0;
    for (int i = reduce_start_idx + tid_in_vec; i < reduce_end_idx; i += vec_size)
    {
      sum += smem[i];
    }
    sum = warpReduceSum<vec_size>(sum);
    // store value
    if (tid_in_vec == 0)
    {
      atomicAdd(y + reduce_row_id, sum);
    }
  }
}


template <int THREADS_PER_BLOCK, int VECTOR_SIZE>
__device__ __forceinline__ void
reduce_oneRow_in_vector_L(int NNZ_PER_BLOCK, const int n_reduce_rows_num, const int tid_in_block, const int block_id,
                          const int reduceStartRowId, const int reduceEndRowId,
                          const int *__restrict__ row_ptr, const valT *__restrict__ smem, valT *__restrict__ y)
{
  // use `vec_num` vectors, each vector can process reduction of one row by involving `vec_size` threads.
  const int vec_size = VECTOR_SIZE;
  const int vec_num = THREADS_PER_BLOCK / vec_size;
  const int vec_id = tid_in_block / vec_size;
  const int tid_in_vec = tid_in_block & (vec_size - 1);
  const int warp_lane_id = tid_in_block & 31;

  int reduce_row_id = reduceStartRowId + vec_id;
  for (; reduce_row_id < reduceEndRowId; reduce_row_id += vec_num)
  {
    const int reduce_start_idx = max((int)0, row_ptr[reduce_row_id] - block_id * NNZ_PER_BLOCK);
    const int reduce_end_idx = min((int)NNZ_PER_BLOCK, row_ptr[reduce_row_id + 1] - block_id * NNZ_PER_BLOCK);
    // reduce LDS via vectors.
    valT sum = 0;
    for (int i = reduce_start_idx + tid_in_vec; i < reduce_end_idx; i += vec_size)
    {
      sum += smem[i];
    }
    // sum = warpReduceSum<vec_size>(sum);
    for (int offset = 16; offset > 0; offset >>= 1)
    {
      sum += __shfl_down_sync(0xffffffff, sum, offset);
    }
    // store value
    if (warp_lane_id == 0)
    {
      atomicAdd(y + reduce_row_id, sum);
    }
  }
}



// optimized for 2 atomic add


template <int THREADS_PER_BLOCK>
__device__ __forceinline__ void reduce_oneRow_in_block_optimized(int NNZ_PER_BLOCK, const int tid_in_block, const int block_id,
                                                                 const int reduceStartRowId,
                                                                 const int *__restrict__ row_ptr,
                                                                 const valT *__restrict__ smem, valT *__restrict__ y)
{
  constexpr int num_warps = THREADS_PER_BLOCK >> 5;
  __shared__ valT warp_results[num_warps];
  const int reduce_start_idx = max((int)0, row_ptr[reduceStartRowId] - block_id * NNZ_PER_BLOCK);
  const int reduce_end_idx = min(NNZ_PER_BLOCK, row_ptr[reduceStartRowId + 1] - block_id * NNZ_PER_BLOCK);

  valT thread_sum = 0;
  for (int j = reduce_start_idx + tid_in_block; j < reduce_end_idx; j += THREADS_PER_BLOCK)
  {
    thread_sum += smem[j];
  }

  for (int offset = 16; offset > 0; offset >>= 1)
  {
    thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
  }
  const int warp_id = tid_in_block >> 5;
  const int lane_id = tid_in_block & 31;
  if (lane_id == 0)
  {
    warp_results[warp_id] = thread_sum;
  }
  __syncthreads();

  if (tid_in_block == 0)
  {
    valT final_sum = 0;
    for (int i = 0; i < num_warps; i++)
    {
      final_sum += warp_results[i];
    }
    atomicAdd(y + reduceStartRowId, final_sum);
  }
}

template <int THREADS_PER_BLOCK>
__device__ __forceinline__ void reduce_oneRow_in_thread_2atomic(int NNZ_PER_BLOCK, const int tid_in_block, const int block_id,
                                                                const int reduceStartRowId, const int reduceEndRowId,
                                                                const int *__restrict__ row_ptr,
                                                                const valT *__restrict__ smem, valT *__restrict__ y)
{
  int reduce_row_id = reduceStartRowId + tid_in_block;
  int nnz_id_before = block_id * NNZ_PER_BLOCK;
  for (; reduce_row_id < reduceEndRowId; reduce_row_id += THREADS_PER_BLOCK)
  {
    valT sum = 0;
    const int reduce_start_idx = max((int)0, row_ptr[reduce_row_id] - nnz_id_before);
    const int reduce_end_idx = min(NNZ_PER_BLOCK, row_ptr[reduce_row_id + 1] - nnz_id_before);
    for (int i = reduce_start_idx; i < reduce_end_idx; i++)
    {
      sum += smem[i];
    }
    // atomicAdd(y + reduce_row_id, sum);
    if (reduce_row_id == reduceStartRowId || reduce_row_id == reduceEndRowId - 1)
    {
      atomicAdd(y + reduce_row_id, sum);
    }
    else
    {
      y[reduce_row_id] += sum;
    }
  }
}

template <int THREADS_PER_BLOCK, int VECTOR_SIZE>
__device__ __forceinline__ void
reduce_oneRow_in_vector_2atomic(int NNZ_PER_BLOCK, const int n_reduce_rows_num, const int tid_in_block, const int block_id,
                                const int reduceStartRowId, const int reduceEndRowId,
                                const int *__restrict__ row_ptr, const valT *__restrict__ smem, valT *__restrict__ y)
{
  // use `vec_num` vectors, each vector can process reduction of one row by involving `vec_size` threads.
  const int vec_size = VECTOR_SIZE;
  const int vec_num = THREADS_PER_BLOCK / vec_size;
  const int vec_id = tid_in_block / vec_size;
  const int tid_in_vec = tid_in_block & (vec_size - 1);

  int reduce_row_id = reduceStartRowId + vec_id;
  for (; reduce_row_id < reduceEndRowId; reduce_row_id += vec_num)
  {
    const int reduce_start_idx = max((int)0, row_ptr[reduce_row_id] - block_id * NNZ_PER_BLOCK);
    const int reduce_end_idx = min((int)NNZ_PER_BLOCK, row_ptr[reduce_row_id + 1] - block_id * NNZ_PER_BLOCK);
    // reduce LDS via vectors.
    valT sum = 0;
    for (int i = reduce_start_idx + tid_in_vec; i < reduce_end_idx; i += vec_size)
    {
      sum += smem[i];
    }
    sum = warpReduceSum<vec_size>(sum);
    // store value
    if (tid_in_vec == 0)
    {
      // atomicAdd(y + reduce_row_id, sum);
      // atomicAdd(y + reduce_row_id, sum);
      if (reduce_row_id == reduceStartRowId || reduce_row_id == reduceEndRowId - 1)
      {
        atomicAdd(y + reduce_row_id, sum);
      }
      else
      {
        y[reduce_row_id] += sum;
      }
    }
  }
}

template <int THREADS_PER_BLOCK>
__global__ void cdspmv_kernel(valT *__restrict__ d_val,
                              int *__restrict__ d_ptr,
                              int *__restrict__ d_cols,
                              int rowA,
                              valT *__restrict__ d_x,
                              valT *__restrict__ d_y,
                              int *__restrict__ startRowPerBlock,
                              int productNnzPerThread,
                              int productNnzPerBlock)
{
  // int NNZ_PER_BLOCK = THREADS_PER_BLOCK * productNnzPerThread;
  extern __shared__ valT middle_s[];
  // __shared__ valT middle_s[NNZ_PER_BLOCK];
  const int last = d_ptr[rowA] - 1;
  int blockNnzStart = productNnzPerBlock * blockIdx.x;

#pragma unroll
  for (int round = 0; round < productNnzPerThread; round++)
  {
    const int sIdx = threadIdx.x + round * THREADS_PER_BLOCK;
    const int gIdx = min(blockNnzStart + sIdx, last);
    middle_s[sIdx] = d_val[gIdx] * __ldg(&d_x[d_cols[gIdx]]);
  }
  __syncthreads();

  const int reduceStartRowId = min(startRowPerBlock[blockIdx.x], rowA);
  int reduceEndRowId = min(startRowPerBlock[blockIdx.x + 1], rowA);
  reduceEndRowId = (reduceEndRowId == 0) ? rowA : reduceEndRowId;
  if (d_ptr[reduceEndRowId] % productNnzPerBlock != 0 || reduceEndRowId == reduceStartRowId)
  {
    reduceEndRowId = min(reduceEndRowId + 1, rowA);
  }

  const int n_reduce_rows_num = reduceEndRowId - reduceStartRowId;
  if (n_reduce_rows_num > 64)
  {
    reduce_oneRow_in_thread_2atomic<THREADS_PER_BLOCK>(productNnzPerBlock, threadIdx.x, blockIdx.x,
                                               reduceStartRowId, reduceEndRowId,
                                               d_ptr, middle_s, d_y);
  }
  else if (n_reduce_rows_num == 1)
  {
    reduce_oneRow_in_block_optimized<THREADS_PER_BLOCK>(productNnzPerBlock, threadIdx.x, blockIdx.x,
                                              reduceStartRowId,
                                              d_ptr, middle_s, d_y);
  }
  else if (n_reduce_rows_num == 2)
  {
    reduce_oneRow_in_vector_L<THREADS_PER_BLOCK, 64>(productNnzPerBlock, n_reduce_rows_num, threadIdx.x, blockIdx.x,
                                                     reduceStartRowId, reduceEndRowId,
                                                     d_ptr, middle_s, d_y);
  }
  else if (n_reduce_rows_num <= 4)
  {
    reduce_oneRow_in_vector_2atomic<THREADS_PER_BLOCK, 32>(productNnzPerBlock, n_reduce_rows_num, threadIdx.x, blockIdx.x,
                                                   reduceStartRowId, reduceEndRowId,
                                                   d_ptr, middle_s, d_y);
  }
  else if (n_reduce_rows_num <= 8)
  {
    reduce_oneRow_in_vector_2atomic<THREADS_PER_BLOCK, 16>(productNnzPerBlock, n_reduce_rows_num, threadIdx.x, blockIdx.x,
                                                   reduceStartRowId, reduceEndRowId,
                                                   d_ptr, middle_s, d_y);
  }
  else if (n_reduce_rows_num <= 16)
  {
    reduce_oneRow_in_vector_2atomic<THREADS_PER_BLOCK, 8>(productNnzPerBlock, n_reduce_rows_num, threadIdx.x, blockIdx.x,
                                                  reduceStartRowId, reduceEndRowId,
                                                  d_ptr, middle_s, d_y);
  }
  else if (n_reduce_rows_num <= 32)
  {
    reduce_oneRow_in_vector_2atomic<THREADS_PER_BLOCK, 4>(productNnzPerBlock, n_reduce_rows_num, threadIdx.x, blockIdx.x,
                                                  reduceStartRowId, reduceEndRowId,
                                                  d_ptr, middle_s, d_y);
  }
  else if (n_reduce_rows_num <= 64)
  {
    reduce_oneRow_in_vector_2atomic<THREADS_PER_BLOCK, 2>(productNnzPerBlock, n_reduce_rows_num, threadIdx.x, blockIdx.x,
                                                  reduceStartRowId, reduceEndRowId,
                                                  d_ptr, middle_s, d_y);
  }
}

void cdspmv(valT *csrVal, int *csrRowPtr, int *csrColInd,
            valT *X_val, valT *Y_val, int rowA, int colA, int nnzA,
            double *cdTime, double *cdPre)
{
  CudaTimer timer;
  valT *d_vecY_accu, *d_vecY_perf, *d_vecX, *d_val;
  int *d_indices, *d_ptr;

  hipMalloc(&d_vecY_accu, sizeof(valT) * rowA);
  hipMalloc(&d_vecY_perf, sizeof(valT) * rowA);
  hipMalloc(&d_vecX, sizeof(valT) * colA);
  hipMalloc(&d_val, sizeof(valT) * nnzA);
  hipMalloc(&d_indices, sizeof(int) * nnzA);
  hipMalloc(&d_ptr, sizeof(int) * (rowA + 2));

  hipMemcpy(d_val, csrVal, sizeof(valT) * nnzA, hipMemcpyHostToDevice);
  hipMemcpy(d_indices, csrColInd, sizeof(int) * nnzA, hipMemcpyHostToDevice);
  hipMemcpy(d_ptr, csrRowPtr, sizeof(int) * (rowA + 2), hipMemcpyHostToDevice);
  hipMemcpy(d_vecX, X_val, sizeof(valT) * colA, hipMemcpyHostToDevice);

  const int THREADS_PER_BLOCK = 128;

#ifdef fp64
  int productNnzPerThread = (nnzA > 200000000) ? 8 : 4;
#else
  int productNnzPerThread = (nnzA > 300000) ? 16 : 4;
#endif
  const int WORK_BLOCKS = nnzA / (productNnzPerThread * THREADS_PER_BLOCK) + ((nnzA % (productNnzPerThread * THREADS_PER_BLOCK) == 0) ? 0 : 1);

  const int startRowPerBlock_len = WORK_BLOCKS + 1;

  int *startRowPerBlock;
  hipMalloc(&startRowPerBlock, sizeof(int) * startRowPerBlock_len);
  hipMemset(startRowPerBlock, 0, sizeof(int) * startRowPerBlock_len);
  timer.start();
  pre_startRowPerBlock<<<divup<uint32_t>(rowA + 1, 128), 128>>>(d_ptr, rowA, startRowPerBlock, productNnzPerThread * THREADS_PER_BLOCK);
  *cdPre = (double)timer.stop();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    printf("pre_startRowPerBlock kernel launch failed: %s\n", hipGetErrorString(err));
  }
  int productNnzPerBlock = THREADS_PER_BLOCK * productNnzPerThread;

  int warm_iter = 200;
  int test_iter = 4000;
  hipMemset(d_vecY_perf, 0.0, sizeof(valT) * rowA);
  for (int i = 0; i < warm_iter; ++i)
  {
    cdspmv_kernel<THREADS_PER_BLOCK><<<(WORK_BLOCKS), (THREADS_PER_BLOCK), productNnzPerBlock * sizeof(valT)>>>(d_val, d_ptr, d_indices, rowA, d_vecX, d_vecY_perf, startRowPerBlock, productNnzPerThread, productNnzPerBlock);
  }
  hipDeviceSynchronize();
  timer.start();
  for (int i = 0; i < test_iter; ++i)
  {
    cdspmv_kernel<THREADS_PER_BLOCK><<<(WORK_BLOCKS), (THREADS_PER_BLOCK), productNnzPerBlock * sizeof(valT)>>>(d_val, d_ptr, d_indices, rowA, d_vecX, d_vecY_perf, startRowPerBlock, productNnzPerThread, productNnzPerBlock);
  }
  float total_loop_time_ms = timer.stop();

  hipMemset(d_vecY_accu, 0.0, sizeof(valT) * rowA);
  cdspmv_kernel<THREADS_PER_BLOCK><<<(WORK_BLOCKS), (THREADS_PER_BLOCK), productNnzPerBlock * sizeof(valT)>>>(d_val, d_ptr, d_indices, rowA, d_vecX, d_vecY_accu, startRowPerBlock, productNnzPerThread, productNnzPerBlock);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR(hipGetLastError());

  *cdTime = (double)total_loop_time_ms / test_iter;
  double cd_gflops = (double)((long)nnzA * 2) / (total_loop_time_ms * 1e6);

  CUDA_CHECK_ERROR(hipMemcpy(Y_val, d_vecY_accu, sizeof(valT) * rowA, hipMemcpyDeviceToHost));
  hipFree(d_vecY_perf);
  hipFree(d_vecY_accu);
  hipFree(d_vecX);
  hipFree(d_val);
  hipFree(d_indices);
  hipFree(d_ptr);
  hipFree(startRowPerBlock);
}
